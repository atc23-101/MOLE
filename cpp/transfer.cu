#include "hip/hip_runtime.h"
#include "data_transfer.h"
#include "hip/hip_fp16.h"

void cuda_gpufp32_to_cpufp32(const float *input, float *output, int size, hipStream_t stream)
{
    hipMemcpyAsync(output, input, size * sizeof(float), hipMemcpyDeviceToHost, stream);
}

void cuda_cpufp32_to_gpufp32(const float *input, float *output, int size, hipStream_t stream)
{
    hipMemcpyAsync(output, input, size * sizeof(float), hipMemcpyHostToDevice, stream);
}
void cuda_gpufp32_to_gpufp32(const float *input, float *output, int size, hipStream_t stream)
{
    hipMemcpyAsync(output, input, size * sizeof(float), hipMemcpyDeviceToDevice, stream);
}

void cuda_gpufp16_to_cpufp16(const __half *input, __half *output, int size, hipStream_t stream)
{
    hipMemcpyAsync(output, input, size * sizeof(__half), hipMemcpyDeviceToHost, stream);
}